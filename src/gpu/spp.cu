#include "hip/hip_runtime.h"
#define USE_CUDA
#include "graph.hh"
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <spdlog/spdlog.h>
#include <thrust/functional.h>

namespace pp {

__global__ auto Relax(const OutEdge *adj_list, uint64_t adj_list_len,
                      uint64_t min_dis, uint64_t *cur_dis, bool *visited)
    -> void {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < adj_list_len) {
    auto to = adj_list[tid].To();
    auto dis = adj_list[tid].Distance();
    if (not visited[to] and dis != inf_dis) {
      cur_dis[to] = thrust::min(cur_dis[to], min_dis + dis);
    }
  }
}

auto ParallalSPP(const Graph &g, id_t vid) -> thrust::host_vector<uint64_t> {
  thrust::device_vector<uint64_t> d_dis(g.Vertices().size(), inf_dis);
  auto d_dis_p = thrust::raw_pointer_cast(d_dis.data());

  thrust::device_vector<bool> d_visited(g.Vertices().size(), false);
  auto d_visited_p = thrust::raw_pointer_cast(d_visited.data());

  const int threads_per_block = 128;
  const dim3 block_dim(threads_per_block, 1, 1);
  const dim3 grid_dim(
      (g.Vertices().size() + threads_per_block - 1) / threads_per_block, 1, 1);

  auto min_begin = thrust::make_zip_iterator(d_dis.begin(), d_visited.begin());
  auto min_end = thrust::make_zip_iterator(d_dis.end(), d_visited.end());
  auto less = [] __device__(thrust::tuple<uint64_t, bool> l,
                            thrust::tuple<uint64_t, bool> r) -> bool {
    auto l_dis = l.get<0>();
    auto r_dis = r.get<0>();
    auto l_visited = l.get<1>();
    auto r_visited = r.get<1>();

    return (l_visited == r_visited) ? (l_dis < r_dis) : not l_visited;
  };

  d_dis[0] = 0;

  auto start = std::chrono::steady_clock::now();
  for (uint64_t i = 0; i < g.Vertices().size(); i++) {
    auto it = thrust::min_element(thrust::device, min_begin, min_end, less);
    auto min_dis = *it.get_iterator_tuple().get<0>();
    auto cur_vid = it - min_begin;
    if (d_visited[cur_vid]) {
      break;
    }
    d_visited[cur_vid] = true;
    auto d_adj_list = g.AdjacentList(cur_vid);
    Relax<<<grid_dim, block_dim>>>(d_adj_list.get<0>(), d_adj_list.get<1>(),
                                   min_dis, d_dis_p, d_visited_p);
  }
  auto end = std::chrono::steady_clock::now();
  spdlog::info("{} ms", std::chrono::duration_cast<std::chrono::milliseconds>(
                            end - start)
                            .count());

  return d_dis;
}

} // namespace pp

#undef USE_CUDA